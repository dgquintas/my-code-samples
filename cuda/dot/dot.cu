// simple stupid dot product example from chapter 5 of CUDA by Example
// as worked by myself from scratch


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000000

#define BPG 16
#define TPB 16


__global__ void dot(float *a, float *b, float *c){
    //accumulate thread result on shared mem (per block)
    __shared__ float cache[TPB];
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    float acum = 0;
    while( tid < N ){
        acum += a[tid] * b[tid];
        tid += gridDim.x * blockDim.x;
    }
    cache[ threadIdx.x ] = acum;

    //need to agregate the result for this block into 
    //a single scalar, from the individual thread results
    // for that, we must be sure all threads have finished
    //barrier here.
    __syncthreads();
    

    for(int i=blockDim.x/2; i > 0; i /= 2){
        tid = threadIdx.x;
        // because all threads are gonna go thru here anyway,
        // but only the i leftmost ones should do sth.
        if( tid < i ){ 
            cache[tid] += cache[tid + i];
        }
        __syncthreads(); // regroup before next iteration
    }

    //block's sum at cache[0]
    //somebody's gotta write it back to the host 
    if( threadIdx.x == 0 ){
        c[blockIdx.x] = cache[0];
    }

}

#define sum_squares(x) ((x*(x+1)*(2*x+1))/6)

int main(){
    float a[N], b[N], c[BPG];
    float *dev_a, *dev_b, *dev_c;

    hipMalloc( (void**)&dev_a, N * sizeof(float) );
    hipMalloc( (void**)&dev_b, N * sizeof(float) );
    hipMalloc( (void**)&dev_c, BPG * sizeof(float) );

    for(int i=0; i < N; i++){
        a[i] = i;
        b[i] = 2*i;
    }

    hipMemcpy( dev_a, a, N*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, N*sizeof(float), hipMemcpyHostToDevice );

    dot<<<BPG, TPB>>>(dev_a, dev_b, dev_c);

    hipMemcpy( c, dev_c, BPG*sizeof(float), hipMemcpyDeviceToHost );


    //sum the results from the individual blocks
    float res=0;
    for(int i=0; i < BPG; i++){
        printf("c[%d] = %f\n", i, c[i]);
        res += c[i];
    }
    
    printf("Result: %f (vs. %f)\n", res, 2*sum_squares((float)(N-1)));


    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );

    return 0;
}


