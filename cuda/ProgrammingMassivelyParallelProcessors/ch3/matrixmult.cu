
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <sstream>
#include <string>
#include <iostream>

#define X 16 //will fail with 23, as sqrt(512) 22.sth and there can only be 512 threads per block
#define Y 16

#define GETCOORDS(row, col) (row) * (Y) + (col)

void MatrixMultHost(float* M, float* N, float* P, int width){

    for(int i=0; i < width; i++){
        for(int j=0; j < width; j++){
           float sum = 0;
           for(int k = 0; k < width; k++){
               sum += M[ GETCOORDS(i,k) ] * N[ GETCOORDS(k,j) ];
           }
           P[ GETCOORDS(i,j) ] = sum;
        }
    } 
}

std::string sprint(float *M){
    std::ostringstream oss;
    for( int i = 0; i < X; i++) {
        for( int j = 0; j < Y; j++) {
            oss << M[ GETCOORDS(i,j) ] << " " ;
        }
        oss << std::endl;
    }
    return oss.str();
}

__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int width){

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float sum = 0.0f;
    for(int i=0; i < width; i++){
        float elemM = Md[ GETCOORDS(tx,i) ];
        float elemN = Nd[ GETCOORDS(i,ty) ];

        sum += elemM * elemN;
    }
    Pd[tx*width + ty] = sum;

}

void MatrixMultDevice(float* M, float* N, float* P, int width){
    int size = width*width*sizeof(float);
    float *Md, *Nd, *Pd;

    //0. reserve memory on device
    hipMalloc( (void**)&Md, size );
    hipMalloc( (void**)&Nd, size );
    hipMalloc( (void**)&Pd, size );

    //1. transfer M and N to device memory
    hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
    hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

    //2. kernel invokation 
    dim3 dimBlock(width, width);
    dim3 dimGrid(1,1);
    MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, width);

    //3. copy P from device to host
    hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);

    //4. Free Md, Nd, Pd
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
}


int main(){
    float *M, *N, *P;

    // allocate M, N, P
    M = (float*)malloc( sizeof(float)*X*Y );
    N = (float*)malloc( sizeof(float)*X*Y );
    P = (float*)malloc( sizeof(float)*X*Y );

    for( int i = 0; i < X; i++) {
        for( int j = 0; j < Y; j++) {
            M[ GETCOORDS(i,j) ] = i+j;
        }
    }
    for( int i = 0; i < X; i++) {
        for( int j = 0; j < Y; j++) {
            N[ GETCOORDS(i,j) ] = i*j;
        }
    }

    // M*N on host
    MatrixMultHost(M,N,P, Y);

    const std::string Phost(sprint(P));
    //std::cout << Phost << std::endl;

    memset(P, 0, sizeof(float)*X*Y );

    // M*N on device
    MatrixMultDevice(M,N,P, Y);

    const std::string Pdev(sprint(P));
    //std::cout << Pdev << std::endl;

    if( Phost != Pdev ){
        std::cout << "FAIL" << std::endl;
    } else{
        std::cout << "WIN" << std::endl;
    }

    // Free M, N, P
    free(M);
    free(N);
    free(P);

    return 0;
}
