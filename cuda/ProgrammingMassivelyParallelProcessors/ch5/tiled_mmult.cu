
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <sstream>
#include <string>
#include <iostream>
#include <stdlib.h>
#include <time.h>

#define X 1600
#define Y 1600

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

#define TILE_WIDTH 16

#define GETCOORDS(row, col) (row) * (Y) + (col)
#define CEIL(x,y) (((x)-1) / (y)) + 1

void MatrixMultHost(float* M, float* N, float* P, int width){
    for(int i=0; i < width; i++){
        for(int j=0; j < width; j++){
           float sum = 0;
           for(int k = 0; k < width; k++){
               sum += M[ GETCOORDS(i,k) ] * N[ GETCOORDS(k,j) ];
           }
           P[ GETCOORDS(i,j) ] = sum;
        }
    } 
}

std::string sprint(float *M){
    std::ostringstream oss;
    for( int i = 0; i < X; i++) {
        for( int j = 0; j < Y; j++) {
            oss << M[ GETCOORDS(i,j) ] << " " ;
        }
        oss << std::endl;
    }
    return oss.str();
}


__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int width){

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int row = by * BLOCK_SIZE_Y + ty;
    int col = bx * BLOCK_SIZE_X + tx;
    if( row >= width || col >= width ) return;

    float res = 0.0f;
    for(int p=0; p < width/TILE_WIDTH; p++){
        Mds[ty][tx] = Md[row*width + (tx + p*TILE_WIDTH)];
        Nds[ty][tx] = Nd[(p*TILE_WIDTH + ty)*width + col ];
        __syncthreads();


        //dot product from shared memory
        for(int k=0; k < TILE_WIDTH; k++){
            res += Mds[ty][k] * Nds[k][tx];
        }  
        __syncthreads();
    }

    Pd[row*width + col] = res;

}

void MatrixMultDevice(float* M, float* N, float* P, int width){
    int size = width*width*sizeof(float);
    float *Md, *Nd, *Pd;

    //0. reserve memory on device
    hipMalloc( (void**)&Md, size );
    hipMalloc( (void**)&Nd, size );
    hipMalloc( (void**)&Pd, size );

    //1. transfer M and N to device memory
    hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
    hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

    hipMemset(Pd, 0, size);

    //2. kernel invokation 
    dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 dimGrid(CEIL(width,BLOCK_SIZE_X), CEIL(width,BLOCK_SIZE_Y));
    printf("Using (%d, %d) blocks of (%d, %d) threads each\n", dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);   

    MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, width);

    //3. copy P from device to host
    hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);

    //4. Free Md, Nd, Pd
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
}


int main(){
    srand(time(NULL));
    float *M, *N, *P;

    // allocate M, N, P
    M = (float*)malloc( sizeof(float)*X*Y );
    N = (float*)malloc( sizeof(float)*X*Y );
    P = (float*)malloc( sizeof(float)*X*Y );

    for( int i = 0; i < X; i++) {
        for( int j = 0; j < Y; j++) {
            M[ GETCOORDS(i,j) ] = rand() % 5;
        }
    }
    for( int i = 0; i < X; i++) {
        for( int j = 0; j < Y; j++) {
            N[ GETCOORDS(i,j) ] = rand() % 5;
        }
    }

    // M*N on host
    MatrixMultHost(M,N,P, Y);

    const std::string Phost(sprint(P));
    std::cout << Phost << std::endl;

    memset(P, 0, sizeof(float)*X*Y );

    // M*N on device
    MatrixMultDevice(M,N,P, Y);

    const std::string Pdev(sprint(P));
    std::cout << Pdev << std::endl;

    if( Phost != Pdev ){
        std::cout << "FAIL" << std::endl;
    } else{
        std::cout << "WIN" << std::endl;
    }

    // Free M, N, P
    free(M);
    free(N);
    free(P);

    return 0;
}
