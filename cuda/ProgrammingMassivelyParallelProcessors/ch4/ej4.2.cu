#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <sstream>
#include <string>
#include <iostream>
#include <stdlib.h>
#include <time.h>

#define X 8
#define Y 8 


#define GETCOORDS(row, col) (row) * (Y) + (col)
#define CEIL(x,y) (((x)-1) / (y)) + 1

std::string sprint(float *M){
    std::ostringstream oss;
    for( int i = 0; i < X; i++) {
        for( int j = 0; j < Y; j++) {
            oss << M[ GETCOORDS(i,j) ] << " " ;
        }
        oss << std::endl;
    }
    return oss.str();
}


__global__ void TransKernel(float* Md, int width){

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ float block[BLOCK_SIZE][BLOCK_SIZE];
    
    int baseIdx = bx * BLOCK_SIZE + tx;
    baseIdx += (by * BLOCK_SIZE + ty) * width;

    block[ty][tx] = Md[baseIdx];
    Md[baseIdx] = block[tx][ty];

}

void MatrixTrans(float* M, int width){
    int size = width*width*sizeof(float);
    float *Md;

    //0. reserve memory on device
    hipMalloc( (void**)&Md, size );

    //1. transfer M and N to device memory
    hipMemcpy(Md, M, size, hipMemcpyHostToDevice);

    //2. kernel invokation 
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(CEIL(width,BLOCK_SIZE), CEIL(width,BLOCK_SIZE));
    printf("Using (%d, %d) blocks of (%d, %d) threads each\n", dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);   

    TransKernel<<<dimGrid, dimBlock>>>(Md, width);

    //3. copy P from device to host
    hipMemcpy(M, Md, size, hipMemcpyDeviceToHost);

    //4. Free Md, Nd, Pd
    hipFree(Md);
}


int main(){
    float *M;

    // allocate M, N, P
    M = (float*)malloc( sizeof(float)*X*Y );

    int k=1;
    for( int i = 0; i < X; i++) {
        for( int j = 0; j < Y; j++) {
            M[ GETCOORDS(i,j) ] = k++;
        }
    }
    const std::string Ms(sprint(M));
    std::cout << Ms << std::endl;

    // M*N on device
    MatrixTrans(M, Y);
    
    const std::string res(sprint(M));
    std::cout << res << std::endl;

    // Free M, N, P
    free(M);

    return 0;
}
