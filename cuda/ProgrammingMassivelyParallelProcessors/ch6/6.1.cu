
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>

#define N (1<<22)
#define BLOCK_SIZE 128

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

__global__ void reduxKernel(int* v, int n, int* res){
    int tx = threadIdx.x;
    int idx = blockIdx.x * (BLOCK_SIZE*2) + tx;
    __shared__ int vs[BLOCK_SIZE*2]; 
    
    // all threads execute the following, as the size of the shared memory is 
    // 2x the number of threads
    int reg = idx < n ? v[idx] : 0;
    if( idx + BLOCK_SIZE < n ){
        reg += v[idx + BLOCK_SIZE];
    }

    //vs[tx] = v[idx] + v[idx + BLOCK_SIZE];
    vs[tx] = reg;
    __syncthreads();

    if( BLOCK_SIZE >= 512 ){
        if( tx < 256 ){ vs[tx] = reg = reg + vs[tx + 256]; __syncthreads(); }
    } 
    if( BLOCK_SIZE >= 256 ){
        if( tx < 128 ){ vs[tx] = reg = reg + vs[tx + 128]; __syncthreads(); }
    }
    if( BLOCK_SIZE >= 128 ){
        if( tx < 64 ){ vs[tx] = reg = reg + vs[tx + 64]; __syncthreads(); }
    }
//    for( int bs = BLOCK_SIZE/2 ; bs > 32; bs >>= 1 ){
//        if( tx < bs ){
//            //accumulating on the register var "reg" saves a read
//            //from shared memory. Instead of doing vs[tx] += vs[tx + bs], 
//            //we just accum on the reg and write it back to shared memory
//            vs[tx] = reg = reg + vs[tx + bs];
//        }
//        __syncthreads();
//    }

    if( tx < 32 ){ //for the last wrap. Threads within a wrap don't need synchronize
        volatile int* synchdShared = vs; //so that the compiler doesn't reorder

        // in case the block size were less than 32 to begin with
        if( BLOCK_SIZE  >= 64 ){
            // means we can "reach out" fully 32 positions to the right of the thread
            // look at the general loop to work out why
            synchdShared[tx] = reg = reg + synchdShared[tx + 32];
        }
        //and of course we are gonna fall-through
        if( BLOCK_SIZE >= 32 ){ 
            synchdShared[tx] = reg = reg + synchdShared[tx + 16];
        }
        if( BLOCK_SIZE >= 16 ){
            synchdShared[tx] = reg = reg + synchdShared[tx + 8];
        }
        if( BLOCK_SIZE >= 8 ){
            synchdShared[tx] = reg = reg + synchdShared[tx + 4];
        }
        if( BLOCK_SIZE >= 4 ){
            synchdShared[tx] = reg = reg + synchdShared[tx + 2];
        }
        if( BLOCK_SIZE >= 2 ){
            synchdShared[tx] = reg = reg + synchdShared[tx + 1];
        }

    }

    //interesting: without this if, the kernel execution takes 5x the time! Probably
    //due the memory conflicts created by _ALL_ the threads trying to write to the same
    //memory location "at once"
    if(tx == 0){ 
       res[blockIdx.x] = vs[0];
    }

}

int hostRedux(int* v, int n){
    int res = 0;
    for(int i=0; i < n; i++){
        res += v[i];
    }
    return res;
}

void deviceRedux(int* v, int n, int* res){
    // reserve memory on device
    int *vd; 
    int *resd;
 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time; 

    const int numBlocks = (N/BLOCK_SIZE)/2;

    HANDLE_ERROR( hipMalloc((void**)&vd, n*sizeof(int)) );
    HANDLE_ERROR( hipMalloc((void**)&resd, numBlocks*sizeof(int)) );

    hipEventRecord( start, 0 ); 
    // transfer v to vd
    HANDLE_ERROR( hipMemcpy(vd, v, n*sizeof(int), hipMemcpyHostToDevice) );
    hipEventRecord( stop, 0 );
    
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );
    printf("memcpy in time: %f ms\n", time);

 
    hipEventRecord( start, 0 );
    // invoke kernel
    reduxKernel<<<numBlocks,BLOCK_SIZE>>>(vd,n,resd);
    hipEventRecord( stop, 0 );
    
    hipEventSynchronize( stop );

    hipEventElapsedTime( &time, start, stop );

    printf("kernel time: %f ms. ", time);
    printf("Bandwidth: %f GB/s\n", (N*sizeof(int)/1e6)/time);

    hipEventRecord( start, 0 );  
    // copy results back to host
    HANDLE_ERROR( hipMemcpy(res, resd, numBlocks*sizeof(int), hipMemcpyDeviceToHost) );
    hipEventRecord( stop, 0 );
    
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );
    printf("memcpy out time: %f ms\n", time);
 
 
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //free vd
    hipFree(vd);
    hipFree(resd);
}

     /* Subtract the `struct timeval' values X and Y,
        storing the result in RESULT.
        Return 1 if the difference is negative, otherwise 0.  */
     
int
timeval_subtract (struct timeval* result, struct timeval*x, struct timeval*y)
{
  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (x->tv_usec - y->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result->tv_sec = x->tv_sec - y->tv_sec;
  result->tv_usec = x->tv_usec - y->tv_usec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}


int main(){
    int* v = (int*)malloc( sizeof(int) * N);
    for(int i=0; i < N; i++){
        v[i] = rand() % 10;
        //v[i] =  i;
    }

    int h;
    struct timeval tv_start;
    struct timeval tv_stop;
    struct timeval tv_diff;
    gettimeofday(&tv_start, 0);
    h=hostRedux(v,N);
    gettimeofday(&tv_stop, 0);
    timeval_subtract(&tv_diff, &tv_stop, &tv_start);
    printf("host: %d\n", h);
    float msdiff = tv_diff.tv_sec*1000 + tv_diff.tv_usec/1000.0;
    printf("CPU time: %.3f ms\n", msdiff);

    const int numBlocks = (N/BLOCK_SIZE)/2;
    int *res = (int*)malloc( sizeof(int) * numBlocks);
    int d;

    hipEvent_t start, stop;
    float time; 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );

    deviceRedux(v,N, res);
    d=0;
    for(int i=0; i < numBlocks; i++){
       // printf("%d\n", res[i]);
        d+=res[i];
    }
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );

    hipEventElapsedTime( &time, start, stop );
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("CUDA time: %f ms\n", time);


    printf("device: %d\n", d);

    free(v);
    free(res);
    return !(h==d);

}
